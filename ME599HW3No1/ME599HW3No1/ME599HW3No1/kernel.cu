#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
#define TPB 32
#define ATOMIC 1 // 0 for non-atomic addition

__global__
void dotKernel(float *d_result, float *d_array_a, float *d_array_b, int n) {
	__shared__ float s_product[TPB];
	const int idx = threadIdx.x + blockDim.x * blockIdx.x;
	const int s_idx = threadIdx.x;
	
	if (idx >= n) {
		s_product[s_idx] = 0;
	}
	else {
	s_product[s_idx] = d_array_a[idx] * d_array_b[idx];
	}
	__syncthreads();


	// shared memory atomicAdd code 
	if (s_idx == 0) {
		float blockSum = 0.0;
		for (int j = 0; j < blockDim.x; ++j) {
			blockSum += s_product[j];
		}
		// Try each of two versions of adding to the accumulator
		if (ATOMIC) {
			atomicAdd(d_result, blockSum);
		}
		else {
			*d_result += blockSum;
		}
	}
}


void dotProduct(float *result, float *array_a, float *array_b, int n) {
	float *d_result;
	float *d_array_a;
	float *d_array_b;
	
	//create event variable for timing
	hipEvent_t startKernel, stopKernel;
	hipEventCreate(&startKernel);
	hipEventCreate(&stopKernel);

	// Allocate memory for device arrays
	hipMalloc(&d_result, sizeof(float));
	hipMalloc(&d_array_a, n*sizeof(float));
	hipMalloc(&d_array_b, n*sizeof(float));

	// Copy inout from host to device 
	hipMemset(d_result, 0.0, sizeof(float));
	hipMemcpy(d_array_a, array_a, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_array_b, array_b, n*sizeof(float), hipMemcpyHostToDevice);
	
	// set shared memory size in byte 
	const size_t smemSize = TPB*sizeof(float);

	// Launch kernel to compute and store values 
	hipEventRecord(startKernel);
	dotKernel << <(n+TPB-1)/TPB, TPB, smemSize >> >(d_result, d_array_a, d_array_b, n);
	hipEventRecord(stopKernel);
	hipMemcpy(result, d_result, sizeof(float), hipMemcpyDeviceToHost);
	// Ensure times events have stoped 
	hipEventSynchronize(stopKernel);

	// convert event records to time and output 
	float kernelTimeInMs = 0;
	hipEventElapsedTime(&kernelTimeInMs, startKernel, stopKernel);
	printf("Kernel time with share memory(ms): %f\n\n", kernelTimeInMs);

	hipFree(d_result);
	hipFree(d_array_a);
	hipFree(d_array_b);
}

__global__
void dotNoSKernel(float *d_result_NoS, float *d_array_a, float *d_array_b, int n) {
	const int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= n) return;
	const int s_idx = threadIdx.x;

	__syncthreads();

	// atomicAdd without shared memory 
	atomicAdd(d_result_NoS, d_array_a[idx] * d_array_b[idx]);

}

void dotProductNoS(float *result_NoS, float *array_a, float *array_b, int n) {
	float *d_result_NoS;
	float *d_array_a;
	float *d_array_b;

	//create event variable for timing
	hipEvent_t startKernelNoS, stopKernelNoS;
	hipEventCreate(&startKernelNoS);
	hipEventCreate(&stopKernelNoS);

	// Allocate memory for device arrays
	hipMalloc(&d_result_NoS, sizeof(float));
	hipMalloc(&d_array_a, n*sizeof(float));
	hipMalloc(&d_array_b, n*sizeof(float));

	// Copy inout from host to device 
	hipMemset(d_result_NoS, 0.0, sizeof(float));
	hipMemcpy(d_array_a, array_a, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_array_b, array_b, n*sizeof(float), hipMemcpyHostToDevice);

	// set shared memory size in byte 

	// Launch kernel to compute and store values 
	hipEventRecord(startKernelNoS);
	dotNoSKernel << <(n + TPB - 1) / TPB, TPB >> >(d_result_NoS, d_array_a, d_array_b, n);
	hipEventRecord(stopKernelNoS);
	hipMemcpy(result_NoS, d_result_NoS, sizeof(float), hipMemcpyDeviceToHost);
	// Ensure times events have stoped 
	hipEventSynchronize(stopKernelNoS);

	// convert event records to time and output 
	float kernelTimeInMsNoS = 0;
	hipEventElapsedTime(&kernelTimeInMsNoS, startKernelNoS, stopKernelNoS);
	printf("Kernel time without shared memory(ms): %f\n\n", kernelTimeInMsNoS);

	hipFree(d_result_NoS);
	hipFree(d_array_a);
	hipFree(d_array_b);
}

